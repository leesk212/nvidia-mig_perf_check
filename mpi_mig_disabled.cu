#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mpi.h>
#include <stdio.h>

#define BLOCK_SIZE 16

__global__ void addition(float *left, float *right, float *res, int dim) {

    int i,j;
    float temp = 0;

    __shared__ float Left_shared_t [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Right_shared_t[BLOCK_SIZE][BLOCK_SIZE];

    // Row i of matrix left
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;


    for (int tileNUM = 0; tileNUM < gridDim.x; tileNUM++) {

        // Column j of matrix left
        j = tileNUM * BLOCK_SIZE + threadIdx.x;
        i = tileNUM * BLOCK_SIZE + threadIdx.y;
        // Load left[i][j] to shared mem

        Left_shared_t[threadIdx.y][threadIdx.x] = left[row * dim + j];// Coalesced access
        // Load right[i][j] to shared mem

        Right_shared_t[threadIdx.y][threadIdx.x] = right[i * dim + col]; // Coalesced access
        // Synchronize before computation
        __syncthreads();

        // Accumulate one tile of res from tiles of left and right in shared mem
        for (int k = 0; k < BLOCK_SIZE; k++) {

            temp += Left_shared_t[threadIdx.y][k] * Right_shared_t[k][threadIdx.x]; //no shared memory bank conflict
        }
        // Synchronize
        __syncthreads();
    }
    // Store accumulated value to res
    res[row * dim + col] = temp;
}
__global__ void multiply(float *left, float *right, float *res, int dim) {

    int i,j;
    float temp = 0;

    __shared__ float Left_shared_t [BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Right_shared_t[BLOCK_SIZE][BLOCK_SIZE];

    // Row i of matrix left
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;


    for (int tileNUM = 0; tileNUM < gridDim.x; tileNUM++) {

        // Column j of matrix left
        j = tileNUM * BLOCK_SIZE + threadIdx.x;
        i = tileNUM * BLOCK_SIZE + threadIdx.y;
        // Load left[i][j] to shared mem

        Left_shared_t[threadIdx.y][threadIdx.x] = left[row * dim + j];// Coalesced access
        // Load right[i][j] to shared mem

        Right_shared_t[threadIdx.y][threadIdx.x] = right[i * dim + col]; // Coalesced access
        // Synchronize before computation
        // Accumulate one tile of res from tiles of left and right in shared mem
        for (int k = 0; k < BLOCK_SIZE; k++) {

            temp += Left_shared_t[threadIdx.y][k] * Right_shared_t[k][threadIdx.x]; //no shared memory bank conflict
        }
        // Synchronize
        __syncthreads();
    }
    // Store accumulated value to res
    res[row * dim + col] = temp;
}
int main(int argc, char* argv[]){

        float *left1, *right1, *left2, *right2, *res1, *res2;
        float *left1_d, *right1_d, *left2_d, *right2_d, *res1_d, *res2_d;
        int dummy = 0;

        int width = atoi(argv[1]);
        //int width = 8196;
        int size = width * width * sizeof(float);

        hipHostMalloc((void **)&left1, size);
        hipHostMalloc((void **)&left2, size);
        hipHostMalloc((void **)&right1, size);
        hipHostMalloc((void **)&right2, size);
        hipHostMalloc((void **)&res1, size);
        hipHostMalloc((void **)&res2, size);

        for(int i = 0; i < width; i++){
                for(int j = 0; j < width; j++){
                        dummy = width * i + j;
                        left1[dummy] = sinf(dummy);
                        right2[dummy] = cosf(dummy);
                }
        }

        for(int i = 0; i < width; i++){
                for(int j = 0; j < width; j++){
                        dummy = width * i + j;
                        right1[dummy] = cosf(dummy);
                        left2[dummy] = sinf(dummy);
                }
        }

        hipMalloc((void **)&left1_d, size);
        hipMalloc((void **)&left2_d, size);
        hipMalloc((void **)&right1_d, size);
        hipMalloc((void **)&right2_d, size);
        hipMalloc((void **)&res1_d, size);
        hipMalloc((void **)&res2_d, size);

        hipMemcpy(left1_d, left1, size, hipMemcpyHostToDevice);
        hipMemcpy(left2_d, left2, size, hipMemcpyHostToDevice);
        hipMemcpy(right1_d, right1, size, hipMemcpyHostToDevice);
        hipMemcpy(right2_d, right2, size, hipMemcpyHostToDevice);
        hipMemcpy(res1_d, res1, size, hipMemcpyHostToDevice);
        hipMemcpy(res2_d, res2, size, hipMemcpyHostToDevice);

        dim3 Block_dim(BLOCK_SIZE, BLOCK_SIZE);
        dim3 Grid_dim(width / BLOCK_SIZE, width / BLOCK_SIZE);

                                                          
	        MPI_Init(NULL, NULL);

        int world_size;
        MPI_Comm_size(MPI_COMM_WORLD, &world_size);
        int world_rank;
        MPI_Comm_rank(MPI_COMM_WORLD, &world_rank);

        if(world_rank == 0){

                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

                hipEventRecord(start,0);

                multiply<<<Grid_dim, Block_dim>>>(left1_d, right1_d, res1_d, width);

                hipEventRecord(stop,0);
                hipEventSynchronize(stop);

                hipDeviceSynchronize();
                hipMemcpy(res1, res1_d, size, hipMemcpyDeviceToHost);

                float et = 0;
                hipEventElapsedTime(&et, start, stop);

                hipEventDestroy(start);
                hipEventDestroy(stop);
                printf("%f\n", et);


        }
        else{

                hipEvent_t start, stop;
                hipEventCreate(&start);
                hipEventCreate(&stop);

                hipEventRecord(start,0);

                //addition<<<Grid_dim, Block_dim>>>(left2_d, right2_d, res2_d, width);
                multiply<<<Grid_dim, Block_dim>>>(left1_d, right1_d, res2_d, width);

                hipEventRecord(stop,0);
                hipEventSynchronize(stop);

                hipDeviceSynchronize();
                hipMemcpy(res2, res2_d, size, hipMemcpyDeviceToHost);

                float et = 0;
                hipEventElapsedTime(&et, start, stop);

                hipEventDestroy(start);
                hipEventDestroy(stop);
                printf("%f\n", et);


        }

        MPI_Finalize();

        hipHostFree(left1);
        hipHostFree(left2);
        hipHostFree(right1);
        hipHostFree(right2);
        hipHostFree(res1);
        hipHostFree(res2);
	
        hipFree(left1_d);
        hipFree(left2_d);
        hipFree(right1_d);
        hipFree(right2_d);
        hipFree(res1_d);
        hipFree(res2_d);



	return 0;
}
